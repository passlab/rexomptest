#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "xomp_cuda_lib.cu" 
#include "xomp_cuda_lib_inlined.cu" 
#include "omptarget.h" 
#include "rex_kmp.h" 
int main(int ,char **);
#ifdef __cplusplus
extern "C" {
#endif
__device__ int32_t omptarget_device_environment;
__device__ char OUT__1__9009__main__7__kernel___exec_mode = 0;

extern struct __tgt_bin_desc* register_cubin(char*, void*, void*);
__global__ void OUT__1__9009__main__7__kernel__()
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0,9,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index,9,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      printf("Test.\n");
    }
}

void OUT__1__9009__main__7__()
{
/* Launch CUDA kernel ... */
  int _threads_per_block_ = 1024;
  int _num_blocks_ = 256;
  void *__entry_ptr = (void *)(&OUT__1__9009__main__7__);
  char __cuda_entry[] = "OUT__1__9009__main__7__kernel__";
  int64_t __device_id = 0;
  struct __tgt_offload_entry __omp_offload_entry = {((void *)(&OUT__1__9009__main__7__)), __cuda_entry, 0, 0, 0};
  struct __tgt_offload_entry *__start_omp_offloading_entries = &__omp_offload_entry;
  struct __tgt_offload_entry *__stop_omp_offloading_entries = &__start_omp_offloading_entries[1];
  char cuda_entry_name[] = "rex_lib_target.cubin";
  struct __tgt_bin_desc *bin_desc = register_cubin(cuda_entry_name,__start_omp_offloading_entries,__stop_omp_offloading_entries);
  void *__host_ptr = __entry_ptr;
  void *__args_base[] = {};
  void *__args[] = {};
  int64_t __arg_sizes[] = {};
  int64_t __arg_types[] = {};
  int32_t __arg_num = 0;
  __tgt_target_teams(__device_id,__host_ptr,__arg_num,__args_base,__args,__arg_sizes,__arg_types,_threads_per_block_,_num_blocks_);
  __tgt_unregister_lib(bin_desc);
}
#ifdef __cplusplus
}
#endif
