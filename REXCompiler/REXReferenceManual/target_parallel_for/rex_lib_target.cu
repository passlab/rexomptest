#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "xomp_cuda_lib.cu" 
#include "xomp_cuda_lib_inlined.cu" 
int main(int ,char **);
#ifdef __cplusplus
extern "C" {
#endif
__device__ int32_t omptarget_device_environment;
__device__ char OUT__1__9009__main__13__kernel___exec_mode = 0;

__global__ void OUT__1__9009__main__13__kernel__()
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0,5,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index,5,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      printf("Test 2.\n");
    }
}
__device__ char OUT__2__9009__main__7__kernel___exec_mode = 0;

__global__ void OUT__2__9009__main__7__kernel__()
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0,1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index,1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      printf("Test 1.\n");
    }
}
#ifdef __cplusplus
}
#endif
