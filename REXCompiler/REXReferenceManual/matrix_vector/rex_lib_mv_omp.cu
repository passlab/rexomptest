#include "hip/hip_runtime.h"
/*
 * matrix vector
 * A[N][N] * B[N] = C[N]
 *
 */
#include <stdio.h>
#include <stdlib.h>
//#include <math.h>
#include <sys/time.h>
#include <string.h>
#define TEAM_NUM 1024
#define TEAM_SIZE 1024
/* read timer in ms */
#include "rex_nvidia.h" 

static double read_timer_ms()
{
  struct timeval timer;
  gettimeofday(&timer,((void *)0));
  return ((double )timer . tv_sec) * 1000.0 + ((double )timer . tv_usec) / 1000.0;
}
#define REAL float
void init(int ,float *);
/*
REAL check(REAL*A, REAL*B, int n)
{
    int i;
    REAL diffsum =0.0, sum = 0.0;
    for (i = 0; i < n; i++) {
        diffsum += fabs(A[i] - B[i]);
        sum += fabs(B[i]);
    }
    return diffsum;
}
*/
void matvec_omp_cpu(int ,float *,float *,float *);
void matvec_omp_gpu(int ,float *,float *,float *);
int main(int ,char *[]);
#ifdef __cplusplus
extern "C" {
#endif
__device__ char OUT__1__4980__matvec_omp_gpu__63__kernel___exec_mode = 0;

__global__ void OUT__1__4980__matvec_omp_gpu__63__kernel__(int *Np__,int *ip__,int *jp__,float *_dev_A,float *_dev_B,float *_dev_C)
{
  int _p_i;
  int _p_j;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *Np__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *Np__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      float temp = (float )0.0;
      for (_p_j = 0; _p_j <  *Np__; _p_j++) 
        temp += _dev_A[_p_i *  *Np__ + _p_j - 0] * _dev_B[_p_j - 0];
      _dev_C[_p_i - 0] = temp;
    }
}
__device__ char OUT__2__4980__matvec_omp_gpu__62__kernel___exec_mode = 0;

__global__ void OUT__2__4980__matvec_omp_gpu__62__kernel__(int *Np__,int *ip__,int *jp__,float *_dev_A,float *_dev_B,float *_dev_C)
{
{
/* Launch CUDA kernel ... */
    int _threads_per_block_ = 1024;
    int _num_blocks_ = 1;
    OUT__1__4980__matvec_omp_gpu__63__kernel__<<<_num_blocks_,_threads_per_block_>>>(Np__,ip__,jp__,_dev_A,_dev_B,_dev_C);
  }
}
#ifdef __cplusplus
}
#endif
