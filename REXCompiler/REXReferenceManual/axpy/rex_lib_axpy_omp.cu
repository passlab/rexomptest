#include "hip/hip_runtime.h"
// Experimental test input for Accelerator directives
//  simplest scalar*vector operations
// Liao 1/15/2013
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#define TEAM_NUM 1024
#define TEAM_SIZE 1024
#include "rex_nvidia.h" 

static double read_timer_ms()
{
  struct timeval timer;
  gettimeofday(&timer,((void *)0));
  return ((double )timer . tv_sec) * 1000.0 + ((double )timer . tv_usec) / 1000.0;
}
/* change this to do saxpy or daxpy : single precision or double precision*/
#define REAL double
#define VEC_LEN 1024000 // use a fixed number for now
/* zero out the entire vector */
void zero(double *,int );
/* initialize a vector with random floating point numbers */
void init(double *,int );
/* serial version */
void axpy(double *,double *,long ,double );
/* omp version */
void axpy_omp(double *,double *,long ,double );
/* compare two arrays and return percentage of difference */
double check(double *,double *,int );
int main(int ,char *[]);
#ifdef __cplusplus
extern "C" {
#endif
__device__ char OUT__1__7822__axpy_omp__49__kernel___exec_mode = 0;

__global__ void OUT__1__7822__axpy_omp__49__kernel__(long *np__,double *ap__,double *_dev_x,double *_dev_y)
{
  typedef int int64_t;
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *np__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *np__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; ((long )_p_i) <= _dev_upper; _p_i += 1) {
      _dev_y[_p_i] +=  *ap__ * _dev_x[_p_i];
    }
}
#ifdef __cplusplus
}
#endif
