#include "hip/hip_runtime.h"
// Experimental test input for Accelerator directives
//  simplest scalar*vector operations
// Liao 1/15/2013
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
//#include <omp.h>
#define TEAM_NUM 1024
#define TEAM_SIZE 1024
#include "rex_nvidia.h" 

/* change this to do saxpy or daxpy : single precision or double precision*/
#define REAL double
#define VEC_LEN 1024000 //use a fixed number for now
/* zero out the entire vector */
void zero(double *,int );
/* initialize a vector with random floating point numbers */
void init(double *,int );
/*serial version */
void axpy(double *,double *,long ,double );
/* compare two arrays and return percentage of difference */
/*
REAL check(REAL*A, REAL*B, int n)
{
    int i;
    REAL diffsum =0.0, sum = 0.0;
    for (i = 0; i < n; i++) {
        diffsum += fabs(A[i] - B[i]);
        sum += fabs(B[i]);
    }
    return diffsum/sum;
}
*/
int main(int ,char *[]);
#ifdef __cplusplus
extern "C" {
#endif
__device__ char OUT__1__5362__axpy__46__kernel___exec_mode = 0;

__global__ void OUT__1__5362__axpy__46__kernel__(long *np__,double *ap__,int *ip__,double *_dev_x,double *_dev_y)
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *np__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *np__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; ((long )_p_i) <= _dev_upper; _p_i += 1) {
      _dev_y[_p_i - 0] +=  *ap__ * _dev_x[_p_i - 0];
    }
}
__device__ char OUT__2__5362__axpy__45__kernel___exec_mode = 0;

__global__ void OUT__2__5362__axpy__45__kernel__(long *np__,double *ap__,int *ip__,double *_dev_x,double *_dev_y)
{
{
/* Launch CUDA kernel ... */
    int _threads_per_block_ = 1024;
    int _num_blocks_ = 256;
    OUT__1__5362__axpy__46__kernel__<<<_num_blocks_,_threads_per_block_>>>(np__,ap__,ip__,_dev_x,_dev_y);
  }
    hipDeviceSynchronize();
}
#ifdef __cplusplus
}
#endif
