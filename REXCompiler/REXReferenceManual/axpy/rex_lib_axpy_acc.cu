#include "hip/hip_runtime.h"
// Experimental test input for Accelerator directives
//  simplest scalar*vector operations
// Liao 1/15/2013
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
//#include <omp.h>
#define TEAM_NUM 1024
#define TEAM_SIZE 1024
#include "rex_nvidia.h" 

static double read_timer_ms()
{
  struct timeval timer;
  gettimeofday(&timer,((void *)0));
  return ((double )timer . tv_sec) * 1000.0 + ((double )timer . tv_usec) / 1000.0;
}
/* change this to do saxpy or daxpy : single precision or double precision*/
#define REAL double
#define VEC_LEN 1024000 //use a fixed number for now
/* zero out the entire vector */
void zero(double *,int );
/* initialize a vector with random floating point numbers */
void init(double *,int );
/*serial version */
void axpy(double *,double *,long ,double );
/* compare two arrays and return percentage of difference */
/*
REAL check(REAL*A, REAL*B, int n)
{
    int i;
    REAL diffsum =0.0, sum = 0.0;
    for (i = 0; i < n; i++) {
        diffsum += fabs(A[i] - B[i]);
        sum += fabs(B[i]);
    }
    return diffsum/sum;
}
*/
int main(int ,char *[]);
#ifdef __cplusplus
extern "C" {
#endif
__device__ char OUT__1__5325__axpy__45__kernel___exec_mode = 0;

__global__ void OUT__1__5325__axpy__45__kernel__(long *np__,double *ap__,double *_dev_x,double *_dev_y)
{
  int i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *np__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *np__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (i = _dev_lower; ((long )i) <= _dev_upper; i += 1) {
      _dev_y[i] +=  *ap__ * _dev_x[i];
    }
}
#ifdef __cplusplus
}
#endif
