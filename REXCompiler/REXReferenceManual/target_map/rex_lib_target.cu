#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "xomp_cuda_lib.cu" 
#include "xomp_cuda_lib_inlined.cu" 
int main(int ,char **);
#ifdef __cplusplus
extern "C" {
#endif
__device__ int32_t omptarget_device_environment;
__device__ char OUT__1__8060__main__15__kernel___exec_mode = 0;

__global__ void OUT__1__8060__main__15__kernel__(int *bp__,int *cp__,int *dp__)
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0,2,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index,2,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
// 4, 0/random, 8
      printf("Test1 %d, %d, %d.\n", *bp__, *cp__, *dp__);
       *bp__ = 40;
       *cp__ = 60;
       *dp__ = 80;
// 40, 60, 80
      printf("Test2 %d, %d, %d.\n", *bp__, *cp__, *dp__);
    }
}
#ifdef __cplusplus
}
#endif
