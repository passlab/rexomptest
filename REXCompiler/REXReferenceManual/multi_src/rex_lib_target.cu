#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
int main(int ,char **);
#include "rex_nvidia.h"
#ifdef __cplusplus
extern "C" {
#endif
__device__ char OUT__1__9009__main__13__kernel___exec_mode = 0;

__global__ void OUT__1__9009__main__13__kernel__()
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0,5,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index,5,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      printf("Test 2.\n");
    }
}
#ifdef __cplusplus
}
#endif
