#include "hip/hip_runtime.h"
/*
 * Square matrix multiplication
 * A[N][N] * B[N][N] = C[N][N]
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>
#define ALLOWED_DIFF 0.0001
#define TEAM_NUM 1024
#define TEAM_SIZE 1024
/* read timer in ms */
#include "rex_nvidia.h" 

static double read_timer_ms()
{
  struct timeval timer;
  gettimeofday(&timer,((void *)0));
  return ((double )timer . tv_sec) * 1000.0 + ((double )timer . tv_usec) / 1000.0;
}
#define REAL double
void init(int ,double *);
/*
void matmul_omp(int N, REAL *A, REAL *B, REAL *C) {
    int i,j,k;
    REAL temp;
#pragma omp parallel for shared(N,A,B,C) private(i,j,k,temp) collapse(2)
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            temp = 0;
            for (k = 0; k < N; k++) {
                temp += (A[i * N + k] * B[k * N + j]);
            }
            C[i * N + j] = temp;
        }
    }
}
*/
void matmul_omp_target(int ,double *,double *,double *);
int main(int ,char *[]);
#ifdef __cplusplus
extern "C" {
#endif
__device__ char OUT__1__4962__matmul_omp_target__56__kernel___exec_mode = 0;

__global__ void OUT__1__4962__matmul_omp_target__56__kernel__(int *Np__,int *ip__,int *jp__,int *kp__,double *_dev_A,double *_dev_B,double *_dev_C)
{
  int _p_i;
  int _p_j;
  int _p_k;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *Np__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *Np__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      for (_p_j = 0; _p_j <  *Np__; _p_j++) {
        double temp = (double )0;
        for (_p_k = 0; _p_k <  *Np__; _p_k++) {
          temp += _dev_A[_p_i *  *Np__ + _p_k - 0] * _dev_B[_p_k *  *Np__ + _p_j - 0];
        }
        _dev_C[_p_i *  *Np__ + _p_j - 0] = temp;
      }
    }
}
__device__ char OUT__2__4962__matmul_omp_target__55__kernel___exec_mode = 0;

__global__ void OUT__2__4962__matmul_omp_target__55__kernel__(int *Np__,int *ip__,int *jp__,int *kp__,double *_dev_A,double *_dev_B,double *_dev_C)
{
{
/* Launch CUDA kernel ... */
    int _threads_per_block_ = 1024;
    int _num_blocks_ = 1;
    OUT__1__4962__matmul_omp_target__56__kernel__<<<_num_blocks_,_threads_per_block_>>>(Np__,ip__,jp__,kp__,_dev_A,_dev_B,_dev_C);
  }
}
#ifdef __cplusplus
}
#endif
