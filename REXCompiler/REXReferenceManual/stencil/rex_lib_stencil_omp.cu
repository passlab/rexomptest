#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <math.h>
#include <string.h>
#include <sys/time.h>
#define REAL double
#define FILTER_HEIGHT 5
#define FILTER_WIDTH 5
#define TEST 10
#define PROBLEM 256
#define TEAM_NUM 1024
#define TEAM_SIZE 1024
// clang -fopenmp -fopenmp-targets=nvptx64 -Xopenmp-target -march=sm_35 --cuda-path=/usr/local/cuda -O3 -lpthread -fpermissive -msse4.1 stencil_metadirective.c -o stencil.out
// Usage: ./stencil.out <size>
// e.g. ./stencil.out 512
#include "rex_nvidia.h" 
void Convolution(const double *src,double *dst,int width,int height,const float *filter,int flt_width,int flt_height);
void ConvolutionMulti(const double *src,double *dst,int width,int height,const float *filter,int flt_width,int flt_height);
void stencil_omp(const double *src,double *dst,int width,int height,const float *filter,int flt_width,int flt_height);
void stencil_omp_target(const double *src,double *dst,int width,int height,const float *filter,int flt_width,int flt_height);

static double read_timer_ms()
{
  struct timeval timer;
  gettimeofday(&timer,((void *)0));
  return ((double )timer . tv_sec) * 1000.0 + ((double )timer . tv_usec) / 1000.0;
}
void print_array(char *,char *,double *,int ,int );
void initialize(int ,int ,double *);
int main(int ,char *[]);
void stencil_omp_target(const double *,double *,int ,int ,const float *,int ,int );
#ifdef __cplusplus
extern "C" {
#endif
__device__ char OUT__1__6066__stencil_omp_target__109__kernel___exec_mode = 0;

__global__ void OUT__1__6066__stencil_omp_target__109__kernel__(int *widthp__,int *heightp__,int *flt_widthp__,int *flt_heightp__,int *ip__,int *jp__,double *_dev_src,double *_dev_dst,float *_dev_filter)
{
  int _p_i;
  int _p_j;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *heightp__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *heightp__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      for (_p_j = 0; _p_j <  *widthp__; _p_j++) {
        double sum = (double )0;
        int m;
        int n;
        for (n = 0; n <  *flt_widthp__; n++) {
          for (m = 0; m <  *flt_heightp__; m++) {
            int x = _p_j + n -  *flt_widthp__ / 2;
            int y = _p_i + m -  *flt_heightp__ / 2;
            if (x >= 0 && x <  *widthp__ && y >= 0 && y <  *heightp__) {
              int idx = m *  *flt_widthp__ + n;
              sum += _dev_src[y *  *widthp__ + x - 0] * ((double )_dev_filter[idx - 0]);
            }
          }
        }
        _dev_dst[_p_i *  *widthp__ + _p_j - 0] = sum;
      }
    }
}
__device__ char OUT__2__6066__stencil_omp_target__107__kernel___exec_mode = 0;

__global__ void OUT__2__6066__stencil_omp_target__107__kernel__(int *widthp__,int *heightp__,int *flt_widthp__,int *flt_heightp__,int *ip__,int *jp__,double *_dev_src,double *_dev_dst,float *_dev_filter)
{
{
/* Launch CUDA kernel ... */
    int _threads_per_block_ = 1024;
    int _num_blocks_ = 256;
    OUT__1__6066__stencil_omp_target__109__kernel__<<<_num_blocks_,_threads_per_block_>>>(widthp__,heightp__,flt_widthp__,flt_heightp__,ip__,jp__,_dev_src,_dev_dst,_dev_filter);
  }
}
#ifdef __cplusplus
}
#endif
