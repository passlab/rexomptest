#include "hip/hip_runtime.h"
// Experimental test input for Accelerator directives
//  simplest scalar*vector operations
// Liao 1/15/2013
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>
#define NUM_RUNS 10
#include "rex_nvidia.h" 
double read_timer_ms();
/* change this to do saxpy or daxpy : single precision or double precision*/
#define REAL double
#define VEC_LEN 1024000 //use a fixed number for now
/* zero out the entire vector */
void zero(double *,int );
/* initialize a vector with random floating point numbers */
void init(double *,int );
/*serial version */
void axpy(double *,double *,long ,double );
/* compare two arrays and return percentage of difference */
double check(double *,double *,int );
void axpy_ompacc(double *,double *,int ,double );
int main(int ,char *[]);
#ifdef __cplusplus
extern "C" {
#endif
__device__ char OUT__1__9500__axpy_ompacc__75__kernel___exec_mode = 0;

__global__ void OUT__1__9500__axpy_ompacc__75__kernel__(int *np__,double *ap__,int *ip__,double *_dev_x,double *_dev_y)
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *np__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *np__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      _dev_y[_p_i - 0] +=  *ap__ * _dev_x[_p_i - 0];
    }
}
__device__ char OUT__2__9500__axpy_ompacc__70__kernel___exec_mode = 0;

__global__ void OUT__2__9500__axpy_ompacc__70__kernel__(int *np__,double *ap__,int *ip__,double *_dev_x,double *_dev_y)
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *np__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *np__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      _dev_y[_p_i - 0] +=  *ap__ * _dev_x[_p_i - 0];
    }
}
__device__ char OUT__3__9500__axpy_ompacc__67__kernel___exec_mode = 0;

__global__ void OUT__3__9500__axpy_ompacc__67__kernel__(int *np__,double *ap__,int *ip__,double *_dev_x,double *_dev_y)
{
  double _p_a =  *ap__;
  int _p_i =  *ip__;
  double *_p__dev_x = _dev_x;
  double *_p__dev_y = _dev_y;
  int _p_n =  *np__;
  printf("Parallel Kernel 1\n");
{
/* Launch CUDA kernel ... */
    int _threads_per_block_ = 1024;
    int _num_blocks_ = 256;
    OUT__2__9500__axpy_ompacc__70__kernel__<<<_num_blocks_,_threads_per_block_>>>(np__,ap__,ip__,_dev_x,_dev_y);
  }
  printf("Parallel Kernel 2\n");
{
/* Launch CUDA kernel ... */
    int _threads_per_block_ = 1024;
    int _num_blocks_ = 256;
    OUT__1__9500__axpy_ompacc__75__kernel__<<<_num_blocks_,_threads_per_block_>>>(np__,ap__,ip__,_dev_x,_dev_y);
  }
  printf("Serial Kernel\n");
  for (_p_i = 0; _p_i < _p_n; _p_i++) 
    _p__dev_y[_p_i] += _p_a * _p__dev_x[_p_i];
}
#ifdef __cplusplus
}
#endif
