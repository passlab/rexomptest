#include "hip/hip_runtime.h"
// Experimental test input for Accelerator directives
//  simplest scalar*vector operations
// Liao 1/15/2013
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>
#define NUM_RUNS 10
#include "xomp_cuda_lib.cu" 
#include "xomp_cuda_lib_inlined.cu" 
double read_timer_ms();
/* change this to do saxpy or daxpy : single precision or double precision*/
#define REAL double
#define VEC_LEN 1024000 //use a fixed number for now
/* zero out the entire vector */
void zero(double *,int );
/* initialize a vector with random floating point numbers */
void init(double *,int );
/*serial version */
void axpy(double *,double *,long ,double );
/* compare two arrays and return percentage of difference */
double check(double *,double *,int );
void axpy_ompacc(double *,double *,int ,double );
int main(int ,char *[]);
#ifdef __cplusplus
extern "C" {
#endif
__device__ int32_t omptarget_device_environment;
__device__ char OUT__1__7253__axpy_ompacc__68__kernel___exec_mode = 0;

__global__ void OUT__1__7253__axpy_ompacc__68__kernel__(int *np__,double *ap__,double *_dev_x,double *_dev_y)
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0, *np__ - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index, *np__ - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      _dev_y[_p_i - 0] +=  *ap__ * _dev_x[_p_i - 0];
    }
}
#ifdef __cplusplus
}
#endif
