#include "hip/hip_runtime.h"
#include "axpy.h"
#include <hipblas.h>
#include <stdio.h>

__global__ 
void
axpy_cudakernel_warmingup(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] += a*x[i];
}

__global__ 
void
axpy_cudakernel_1perThread(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] += a*x[i];
}

/* block distribution of loop iteration */
__global__ 
void axpy_cudakernel_block(REAL* x, REAL* y, int n, REAL a) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	int total_threads = gridDim.x * blockDim.x;

	int block_size = n / total_threads; //dividable, TODO handle non-dividiable later
	
	int start_index = thread_num * block_size;
	int stop_index = start_index + block_size;
	int i;
        for (i=start_index; i<stop_index; i++) {
		if (i < n) y[i] += a*x[i];
	}
}

/* cyclic distribution of loop distribution */
__global__
void axpy_cudakernel_cyclic(REAL* x, REAL* y, int n, REAL a) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	int total_threads = gridDim.x * blockDim.x;
	
	int i;
	for (i=thread_num; i<n; i+=total_threads) { 
		if (i < n) y[i] += a*x[i];
	}
}

void axpy_cuda(REAL* x, REAL* y, int n, REAL a) {
  REAL *d_x, *d_y;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, n*sizeof(REAL));

  hipMemcpy(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n*sizeof(REAL), hipMemcpyHostToDevice);

  hipblasHandle_t cublasHandle;
  hipblasStatus_t hipblasStatus_t;

  // cuBLAS AXPY
  double elapsed = read_timer_ms();

  hipblasStatus_t = hipblasCreate(&cublasHandle);
  if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    fprintf(stderr, "error %i\n", hipblasStatus_t);

  hipblasStatus_t = hipblasDaxpy(cublasHandle, n, &a, d_x, 1, d_y, 1);

  elapsed = read_timer_ms() - elapsed;
  kernel_time += elapsed;

  hipMemcpy(y, d_y, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
}

